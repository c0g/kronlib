#include "hip/hip_runtime.h"
//
// Created by Thomas Nickson on 05/07/2015.
//

#include "kronlib.h"
#include "iostream"
#include "gtest/gtest.h"
using namespace kronlib;
TEST(Matrix, SetScalarCUDA)
{
    CUDAMatrix<float> mat(2, 3);
    mat = 1;
    for (int r = 0; r < 2; ++r) {
        for (int c = 0; c < 3; ++c) {
            EXPECT_EQ(mat(r, c), 1);
        }
    }
}
TEST(Matrix, SetSeqCUDA)
{
    CUDAMatrix<float> mat(2, 3);
    mat = 1, 2, 3, 4, 5, 6;
    int ans = 1;
    for (int r = 0; r < 2; ++r) {
        for (int c = 0; c < 3; ++c) {
            EXPECT_EQ(mat(r, c), ans);
            ++ans;
        }
    }
}

TEST(Matrix, MinusInplaceCUDA)
{
    CUDAMatrix<float> mat(2, 2);
    mat = 1, 2, 3, 4;
    
    CUDAMatrix<float> mat_minus(2, 2);
    mat_minus = -1, -2, -3, -4;

    mat.negate_inplace();

    EXPECT_EQ(mat, mat_minus);
}

TEST(Matrix, MinusNotInplace)
{
    CUDAMatrix<float> mat(2, 2);
    mat = 1, 2, 3, 4;
    
    CUDAMatrix<float> mat_minus(2, 2);
    mat_minus = -1, -2, -3, -4;

    auto mat_minus2 = -mat;

    EXPECT_EQ(mat_minus2, mat_minus);
}

TEST(Matrix, ExpInplace)
{
    CUDAMatrix<float> mat(2, 2);
    mat = 1, 2, 3, 4;
    
    CUDAMatrix<float> mat_exp(2, 2);
    mat_exp = std::exp(1), std::exp(2), std::exp(3), std::exp(4);

    mat.exp_inplace();

    EXPECT_EQ(mat, mat_exp);
}

TEST(Matrix, ExpNotInplace)
{
    CUDAMatrix<float> mat(2, 2);
    mat = 1, 2, 3, 4;
    
    CUDAMatrix<float> mat_exp(2, 2);
    mat_exp = std::exp(1), std::exp(2), std::exp(3), std::exp(4);

    auto mat_exp2 = exp(mat);

    EXPECT_EQ(mat_exp2, mat_exp);
}

TEST(Matrix, EqualsAssignment)
{
    CUDAMatrix<float> mat(2, 3);
    mat = 1, 2, 3, 4, 5, 6;
    CUDAMatrix<float> mat2 = mat + 1;
    for (int r = 0; r < 2; ++r) {
        for (int c = 0; c < 3; ++c) {
            if ((r == 0) && (c == 0)) {
                EXPECT_EQ(mat(r, c), 1);
            } else if ((r == 0) && (c == 1)) {
                EXPECT_EQ(mat(r, c), 2);
            } else if ((r == 0) && (c == 2)) {
                EXPECT_EQ(mat(r, c), 3);
            } else if ((r == 1) && (c == 0)) {
                EXPECT_EQ(mat(r, c), 4);
            } else if ((r == 1) && (c == 1)) {
                EXPECT_EQ(mat(r, c), 5);
            } else if ((r == 1) && (c == 2)) {
                EXPECT_EQ(mat(r, c), 6);
            }
        }
    }
}

TEST(Matrix, Equality)
{
    CUDAMatrix<float> mat1(2, 3);
    mat1 = 1, 2, 3, 4, 5, 6;
    CUDAMatrix<float> mat2(2, 3);
    mat2 = 1, 2, 3, 4, 5, 6;
    ASSERT_TRUE(mat1 == mat2);
}
TEST(Matrix, Inequality)
{
    CUDAMatrix<float> mat1(2, 3);
    mat1 = 1, 2, 3, 4, 5, 6;
    CUDAMatrix<float> mat2(2, 3);
    mat2 = 1, 2, 3, 4, 5, 7;
    ASSERT_FALSE(mat1 == mat2);
}

TEST(Matrix, TransposeIndexing)
{
    CUDAMatrix<float> mat(20, 30);
    for (int r = 0; r < 20; ++r) {
        for (int c = 0; c < 30; ++c) {
            mat.setat(r, c, r + c);
        }
    }
    auto mat2 = mat.transpose();
    for (int r = 0; r < 20; ++r) {
        for (int c = 0; c < 30; ++c) {
            EXPECT_EQ(mat(r, c), mat2(c, r));
        }
    }
}

TEST(Matrix, Negate)
{
    CUDAMatrix<float> mat(2, 3);
    mat = 1, 2, 3, 4, 5, 6;
    auto mat2 = -mat;
    for (int r = 0; r < 2; ++r) {
        for (int c = 0; c < 3; ++c) {
            EXPECT_EQ(mat(r, c), -mat2(r, c));
        }
    }
}

TEST(Matrix, ScalarAdd)
{
    CUDAMatrix<float> mat(2, 3);
    mat = 1, 2, 3, 4, 5, 6;
    CUDAMatrix<float> mat2 = mat + 1;
    for (int r = 0; r < 2; ++r) {
        for (int c = 0; c < 3; ++c) {
            EXPECT_EQ(mat(r, c) + 1, mat2(r, c));
        }
    }
}

TEST(Matrix, ScalarMinus)
{
    CUDAMatrix<float> mat(2, 3);
    mat = 1, 2, 3, 4, 5, 6;
    CUDAMatrix<float> mat2 = mat - 1;
    for (int r = 0; r < 2; ++r) {
        for (int c = 0; c < 3; ++c) {
            EXPECT_EQ(mat(r, c) - 1, mat2(r, c));
        }
    }
}

TEST(Matrix, MatrixAdd)
{
    CUDAMatrix<float> mat1(2, 3);
    mat1 = 1, 2, 3, 4, 5, 6;
    CUDAMatrix<float> mat2(2, 3);
    mat2 = 0, 1, 2, 3, 4, 5;

    auto mat_sum = mat1 + mat2;

    for (int r = 0; r < 2; ++r) {
        for (int c = 0; c < 3; ++c) {
            EXPECT_EQ(mat1(r, c) + mat2(r, c), mat_sum(r, c));
        }
    }
}

TEST(Matrix, MatrixMinus)
{
    CUDAMatrix<float> mat1(2, 3);
    mat1 = 1, 2, 3, 4, 5, 6;
    CUDAMatrix<float> mat2(2, 3);
    mat2 = 0, 1, 2, 3, 4, 5;

    auto mat_sum = mat1 - mat2;

    for (int r = 0; r < 2; ++r) {
        for (int c = 0; c < 3; ++c) {
            EXPECT_EQ(mat1(r, c) - mat2(r, c), mat_sum(r, c));
        }
    }
}

TEST(Matrix, ScalarMul)
{
    CUDAMatrix<float> mat(2, 3);
    mat = 1, 2, 3, 4, 5, 6;
    CUDAMatrix<float> mat2 = mat * 2;
    for (int r = 0; r < 2; ++r) {
        for (int c = 0; c < 3; ++c) {
            EXPECT_EQ(mat(r, c) * 2, mat2(r, c));
        }
    }
}

TEST(Matrix, ScalarDiv)
{
    CUDAMatrix<float> mat(2, 3);
    mat = 1, 2, 3, 4, 5, 6;
    CUDAMatrix<float> mat2 = mat / 2;
    for (int r = 0; r < 2; ++r) {
        for (int c = 0; c < 3; ++c) {
            EXPECT_EQ(mat(r, c) / 2, mat2(r, c));
        }
    }
}



TEST(Matrix, Hadamard)
{
    CUDAMatrix<float> mat1(2, 3);
    mat1 = 1, 2, 3, 4, 5, 6;
    CUDAMatrix<float> mat2(2, 3);
    mat2 = 0, 1, 2, 3, 4, 5;
    auto mat_hadam = mat1.elemwise_mult(mat2);
    for (int r = 0; r < 2; ++r) {
        for (int c = 0; c < 3; ++c) {
            EXPECT_EQ(mat1(r, c) * mat2(r, c), mat_hadam(r, c));
        }
    }
}
TEST(Matrix, SimpleProduct)
{
    CUDAMatrix<float> mat1(2, 3);
    mat1 = 1, 2, 3, 4, 5, 6;
    CUDAMatrix<float> mat2(3, 2);
    mat2 = 0, 1, 2, 3, 4, 5;

    CUDAMatrix<float> mat_ans12(2, 2);
    mat_ans12 = 16, 22, 34, 49;

    CUDAMatrix<float> mat_ans21(3, 3);
    mat_ans21 = 4, 5, 6, 14, 19, 24, 24, 33, 42;

    auto mat_prod12 = mat1 * mat2;
    auto mat_prod21 = mat2 * mat1;

    EXPECT_EQ(mat_prod12, mat_ans12);
    EXPECT_EQ(mat_prod21, mat_ans21);
}

TEST(Matrix, ProductWithTrans)
{
    CUDAMatrix<float> mat1(2, 3);
    mat1 = 1, 2, 3, 4, 5, 6;
    auto mat2 = mat1.transpose();

    CUDAMatrix<float> mat_ans(2, 2);
    mat_ans = 14, 32, 32, 77;

    auto mat_prod = mat1 * mat2;

    EXPECT_EQ(mat_prod, mat_ans);
}

TEST(Matrix, ProductShape)
{
    CUDAMatrix<float> mat1(2, 3);
    mat1 = 1, 2, 3, 4, 5, 6;

    CUDAMatrix<float> mat2(3, 1);
    mat2 = 1, 2, 3;


    CUDAMatrix<float> mat_ans(2, 1);
    mat_ans = 14, 32;

    auto mat_prod = mat1 * mat2;

    EXPECT_EQ(mat_prod, mat_ans);
    EXPECT_EQ(mat_prod.nR(), mat_ans.nR());
    EXPECT_EQ(mat_prod.nC(), mat_ans.nC());

}

TEST(Matrix, BigProductSingleCUDA)
{
    int M = 200;
    int N = 200;
    int K = 200;
    CUDAMatrix<float> hmat1(M, N);
    CUDAMatrix<float> hmat2(N, K);
    for (int r = 0; r < M; ++r) {
        for (int c = 0; c < N; ++c) {
            hmat1.setat(r, c, (r + c)/1000);
        }
    }
    for (int r = 0; r < N; ++r) {
        for (int c = 0; c < K; ++c) {
            hmat2.setat(r, c, (r + c)/1000);
        }
    }
    // Manual Matrix mult
    CUDAMatrix<float> hmat_ans(M, K);
    hmat_ans = 0;
    for (int r = 0; r < M; ++r) {
        for (int c = 0; c < K; ++c) {
            for (int el = 0; el < N; ++el) {
                hmat_ans.setat(r, c, hmat_ans(r,c) + hmat1(r, el) * hmat2(el, c));
            }
        }
    }
    auto hmat_prod = hmat1 * hmat2;

    EXPECT_EQ(hmat_prod, hmat_ans);
    EXPECT_EQ(hmat_ans.nR(), hmat_prod.nR());
    EXPECT_EQ(hmat_ans.nC(), hmat_prod.nC());

}

TEST(Matrix, Reshape)
{
    CUDAMatrix<float> mat1(2, 3);
    mat1 = 1, 2, 3, 
           4, 5, 6;

    CUDAMatrix<float> mat2(3, 2);
    mat2 = 1, 5,
           4, 3,
           2, 6;

    EXPECT_EQ(mat2.reshape(2, 3), mat1);
}

TEST(Matrix, TransposeReshape)
{
    CUDAMatrix<float> mat(2, 3);
    mat = 1, 3, 4, 5, 7, 9;

    CUDAMatrix<float> ans(2, 3);
    ans = 1, 4, 7, 3, 5, 9;

    auto mat_trans = mat.transpose();
    auto mat_res = mat_trans.reshape(2, 3);

    EXPECT_EQ(ans, mat_res);
}

TEST(Matrix, ReshapeTranspose)
{
    CUDAMatrix<float> mat(2, 3);
    mat = 1, 3, 4, 5, 7, 9;

    CUDAMatrix<float> ans(2, 3);
    ans = 1, 5, 3, 7, 4, 9;

    auto mat_trans = mat.reshape(3, 2).transpose();

    EXPECT_EQ(ans, mat_trans);
}

TEST(Matrix, TransposeReshapeTranspose)
{
    CUDAMatrix<float> mat(2, 3);
    mat = 1, 3, 4, 5, 7, 9;

    CUDAMatrix<float> ans(3, 2);
    ans = 1, 3, 4, 5, 7, 9;

    auto mat_trans = mat.transpose().reshape(2, 3).transpose();

    EXPECT_EQ(ans, mat_trans);
}

TEST(Matrix, ReshapeTransposeReshape)
{
    CUDAMatrix<float> mat(2, 3);
    mat = 1, 3, 4, 5, 7, 9;

    CUDAMatrix<float> ans(2, 3);
    ans = 1, 4, 7, 3, 5 ,9;

    auto mat_trans = mat.reshape(2, 3).transpose().reshape(2, 3);

    EXPECT_EQ(ans, mat_trans);
}

TEST(Matrix, TraceEqual)
{
    CUDAMatrix<float> mat(2, 2);
    mat = 1, 2, 3, 4;
    EXPECT_EQ(mat.trace(), 5);
}
TEST(Matrix, TraceTall)
{
    CUDAMatrix<float> mat(3, 2);
    mat = 1, 2, 3, 4, 5, 6;
    EXPECT_EQ(mat.trace(), 5);
}
TEST(Matrix, TraceWide)
{
    CUDAMatrix<float> mat(2, 3);
    mat = 1, 2, 3, 4, 5, 6;
    EXPECT_EQ(mat.trace(), 6);
}
int main(int argc, char **argv)
{
    std::cout << "Running test" << std::endl;
    ::testing::InitGoogleTest(&argc, argv);

    return RUN_ALL_TESTS();
}
