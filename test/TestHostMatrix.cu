#include "hip/hip_runtime.h"
//
// Created by Thomas Nickson on 05/07/2015.
//

#include "kronlib.h"
#include "iostream"
#include "gtest/gtest.h"
using namespace kronlib;
TEST(Matrix, SetScalarHost)
{
    HostMatrix<float> mat(2, 3);
    mat = 1;
    for (int r = 0; r < 2; ++r) {
        for (int c = 0; c < 3; ++c) {
            EXPECT_EQ(mat(r, c), 1);
        }
    }
}
TEST(Matrix, SetSeqHost)
{
    HostMatrix<float> mat(2, 3);
    mat = 1, 2, 3, 4, 5, 6;
    int ans = 1;
    for (int r = 0; r < 2; ++r) {
        for (int c = 0; c < 3; ++c) {
            EXPECT_EQ(mat(r, c), ans);
            ++ans;
        }
    }
}

TEST(Matrix, MinusInplaceHost)
{
    HostMatrix<float> mat(2, 2);
    mat = 1, 2, 3, 4;
    
    HostMatrix<float> mat_minus(2, 2);
    mat_minus = -1, -2, -3, -4;

    mat.negate_inplace();

    EXPECT_EQ(mat, mat_minus);
}

TEST(Matrix, MinusNotInplace)
{
    HostMatrix<float> mat(2, 2);
    mat = 1, 2, 3, 4;
    
    HostMatrix<float> mat_minus(2, 2);
    mat_minus = -1, -2, -3, -4;

    auto mat_minus2 = -mat;

    EXPECT_EQ(mat_minus2, mat_minus);
}

TEST(Matrix, ExpInplace)
{
    HostMatrix<float> mat(2, 2);
    mat = 1, 2, 3, 4;
    
    HostMatrix<float> mat_exp(2, 2);
    mat_exp = std::exp(1), std::exp(2), std::exp(3), std::exp(4);

    mat.exp_inplace();

    EXPECT_EQ(mat, mat_exp);
}

TEST(Matrix, ExpNotInplace)
{
    HostMatrix<float> mat(2, 2);
    mat = 1, 2, 3, 4;
    
    HostMatrix<float> mat_exp(2, 2);
    mat_exp = std::exp(1), std::exp(2), std::exp(3), std::exp(4);

    auto mat_exp2 = exp(mat);

    EXPECT_EQ(mat_exp2, mat_exp);
}

TEST(Matrix, EqualsAssignment)
{
    HostMatrix<float> mat(2, 3);
    mat = 1, 2, 3, 4, 5, 6;
    HostMatrix<float> mat2 = mat + 1;
    for (int r = 0; r < 2; ++r) {
        for (int c = 0; c < 3; ++c) {
            if ((r == 0) && (c == 0)) {
                EXPECT_EQ(mat(r, c), 1);
            } else if ((r == 0) && (c == 1)) {
                EXPECT_EQ(mat(r, c), 2);
            } else if ((r == 0) && (c == 2)) {
                EXPECT_EQ(mat(r, c), 3);
            } else if ((r == 1) && (c == 0)) {
                EXPECT_EQ(mat(r, c), 4);
            } else if ((r == 1) && (c == 1)) {
                EXPECT_EQ(mat(r, c), 5);
            } else if ((r == 1) && (c == 2)) {
                EXPECT_EQ(mat(r, c), 6);
            }
        }
    }
}

TEST(Matrix, Equality)
{
    HostMatrix<float> mat1(2, 3);
    mat1 = 1, 2, 3, 4, 5, 6;
    HostMatrix<float> mat2(2, 3);
    mat2 = 1, 2, 3, 4, 5, 6;
    ASSERT_TRUE(mat1 == mat2);
}
TEST(Matrix, Inequality)
{
    HostMatrix<float> mat1(2, 3);
    mat1 = 1, 2, 3, 4, 5, 6;
    HostMatrix<float> mat2(2, 3);
    mat2 = 1, 2, 3, 4, 5, 7;
    ASSERT_FALSE(mat1 == mat2);
}

TEST(Matrix, NonSquareTrans)
{
    HostMatrix<float> mat1(2, 3);
    mat1 = 1, 3, 5, 2, 4, 6;
    
    HostMatrix<float> mat2(3, 2);
    mat2 = 1, 2, 3, 4, 5, 6;

    EXPECT_EQ(mat1, mat2.transpose());

}
TEST(Matrix, TransposeIndexing)
{
    HostMatrix<float> mat(20, 30);
    for (int r = 0; r < 20; ++r) {
        for (int c = 0; c < 30; ++c) {
            mat.setat(r, c, r + c);
        }
    }
    auto mat2 = mat.transpose();
    for (int r = 0; r < 20; ++r) {
        for (int c = 0; c < 30; ++c) {
            EXPECT_EQ(mat(r, c), mat2(c, r));
        }
    }
}

TEST(Matrix, Negate)
{
    HostMatrix<float> mat(2, 3);
    mat = 1, 2, 3, 4, 5, 6;
    auto mat2 = -mat;
    for (int r = 0; r < 2; ++r) {
        for (int c = 0; c < 3; ++c) {
            EXPECT_EQ(mat(r, c), -mat2(r, c));
        }
    }
}

TEST(Matrix, ScalarAdd)
{
    HostMatrix<float> mat(2, 3);
    mat = 1, 2, 3, 4, 5, 6;
    HostMatrix<float> mat2 = mat + 1;
    for (int r = 0; r < 2; ++r) {
        for (int c = 0; c < 3; ++c) {
            EXPECT_EQ(mat(r, c) + 1, mat2(r, c));
        }
    }
}

TEST(Matrix, ScalarMinus)
{
    HostMatrix<float> mat(2, 3);
    mat = 1, 2, 3, 4, 5, 6;
    HostMatrix<float> mat2 = mat - 1;
    for (int r = 0; r < 2; ++r) {
        for (int c = 0; c < 3; ++c) {
            EXPECT_EQ(mat(r, c) - 1, mat2(r, c));
        }
    }
}

TEST(Matrix, MatrixAdd)
{
    HostMatrix<float> mat1(2, 3);
    mat1 = 1, 2, 3, 4, 5, 6;
    HostMatrix<float> mat2(2, 3);
    mat2 = 0, 1, 2, 3, 4, 5;

    auto mat_sum = mat1 + mat2;

    for (int r = 0; r < 2; ++r) {
        for (int c = 0; c < 3; ++c) {
            EXPECT_EQ(mat1(r, c) + mat2(r, c), mat_sum(r, c));
        }
    }
}

TEST(Matrix, MatrixMinus)
{
    HostMatrix<float> mat1(2, 3);
    mat1 = 1, 2, 3, 4, 5, 6;
    HostMatrix<float> mat2(2, 3);
    mat2 = 0, 1, 2, 3, 4, 5;

    auto mat_sum = mat1 - mat2;

    for (int r = 0; r < 2; ++r) {
        for (int c = 0; c < 3; ++c) {
            EXPECT_EQ(mat1(r, c) - mat2(r, c), mat_sum(r, c));
        }
    }
}

TEST(Matrix, ScalarMul)
{
    HostMatrix<float> mat(2, 3);
    mat = 1, 2, 3, 4, 5, 6;
    HostMatrix<float> mat2 = mat * 2;
    for (int r = 0; r < 2; ++r) {
        for (int c = 0; c < 3; ++c) {
            EXPECT_EQ(mat(r, c) * 2, mat2(r, c));
        }
    }
}

TEST(Matrix, ScalarDiv)
{
    HostMatrix<float> mat(2, 3);
    mat = 1, 2, 3, 4, 5, 6;
    HostMatrix<float> mat2 = mat / 2;
    for (int r = 0; r < 2; ++r) {
        for (int c = 0; c < 3; ++c) {
            EXPECT_EQ(mat(r, c) / 2, mat2(r, c));
        }
    }
}



TEST(Matrix, Hadamard)
{
    HostMatrix<float> mat1(2, 3);
    mat1 = 1, 2, 3, 4, 5, 6;
    HostMatrix<float> mat2(2, 3);
    mat2 = 0, 1, 2, 3, 4, 5;
    auto mat_hadam = mat1.elemwise_mult(mat2);
    for (int r = 0; r < 2; ++r) {
        for (int c = 0; c < 3; ++c) {
            EXPECT_EQ(mat1(r, c) * mat2(r, c), mat_hadam(r, c));
        }
    }
}
TEST(Matrix, SimpleProduct)
{
    HostMatrix<float> mat1(2, 3);
    mat1 = 1, 2, 3, 4, 5, 6;
    HostMatrix<float> mat2(3, 2);
    mat2 = 0, 1, 2, 3, 4, 5;

    HostMatrix<float> mat_ans12(2, 2);
    mat_ans12 = 16, 22, 34, 49;

    HostMatrix<float> mat_ans21(3, 3);
    mat_ans21 = 4, 5, 6, 14, 19, 24, 24, 33, 42;

    auto mat_prod12 = mat1 * mat2;
    auto mat_prod21 = mat2 * mat1;

    EXPECT_EQ(mat_prod12, mat_ans12);
    EXPECT_EQ(mat_prod21, mat_ans21);
}

TEST(Matrix, ProductWithTrans)
{
    HostMatrix<float> mat1(2, 3);
    mat1 = 1, 2, 3, 4, 5, 6;
    auto mat2 = mat1.transpose();

    HostMatrix<float> mat_ans(2, 2);
    mat_ans = 14, 32, 32, 77;

    auto mat_prod = mat1 * mat2;

    EXPECT_EQ(mat_prod, mat_ans);
}

TEST(Matrix, ProductShape)
{
    HostMatrix<float> mat1(2, 3);
    mat1 = 1, 2, 3, 4, 5, 6;

    HostMatrix<float> mat2(3, 1);
    mat2 = 1, 2, 3;


    HostMatrix<float> mat_ans(2, 1);
    mat_ans = 14, 32;

    auto mat_prod = mat1 * mat2;

    EXPECT_EQ(mat_prod, mat_ans);
    EXPECT_EQ(mat_prod.nR(), mat_ans.nR());
    EXPECT_EQ(mat_prod.nC(), mat_ans.nC());

}

TEST(Matrix, ComplexProductOne)
{
    HostMatrix<float> mat1(3, 2);
    mat1 = 1, 2, 3, 4, 5, 6;

    HostMatrix<float> mat2(3, 1);
    mat2 = 1, 2, 3;
    
    auto mat_ans = mat1.transpose().dot(mat2);

    EXPECT_EQ(mat1.dot(Trans, mat2, None), mat_ans);

}
TEST(Matrix, ComplexProductTwo)
{

    HostMatrix<float> mat1(1, 3);
    mat1 = 1, 2, 3;

    HostMatrix<float> mat2(2, 3);
    mat2 = 1, 2, 3, 4, 5, 6;
    
    auto mat_ans = mat1.dot(mat2.transpose());

    EXPECT_EQ(mat1.dot(None, mat2, Trans), mat_ans);

}
TEST(Matrix, ComplexProductThree)
{

    HostMatrix<float> mat1(1, 3);
    mat1 = 1, 2, 3;

    HostMatrix<float> mat2(3, 2);
    mat2 = 1, 2, 3, 4, 5, 6;
    auto mat_ans = mat1.dot(mat2.reshape(2, 3).transpose());

    EXPECT_EQ(mat1.dot(1, 3, 2, None, mat2, Trans), mat_ans);
}

TEST(Matrix, ComplexProductFour)
{

    HostMatrix<float> mat1(2, 3);
    mat1 = 1, 2, 3, 4, 5, 6;

    HostMatrix<float> mat2(3, 2);
    mat2 = 1, 2, 3, 4, 5, 6;
    auto mat_ans = mat1.reshape(3, 2).transpose() * mat2.reshape(2, 3).transpose();

    EXPECT_EQ(mat1.dot(2, 3, 2, Trans, mat2, Trans), mat_ans);
}

TEST(Matrix, BigProductSingleHost)
{
    int M = 200;
    int N = 200;
    int K = 200;
    HostMatrix<float> hmat1(M, N);
    HostMatrix<float> hmat2(N, K);
    for (int r = 0; r < M; ++r) {
        for (int c = 0; c < N; ++c) {
            hmat1.setat(r, c, (r + c)/1000);
        }
    }
    for (int r = 0; r < N; ++r) {
        for (int c = 0; c < K; ++c) {
            hmat2.setat(r, c, (r + c)/1000);
        }
    }
    // Manual Matrix mult
    HostMatrix<float> hmat_ans(M, K);
    hmat_ans = 0;
    for (int r = 0; r < M; ++r) {
        for (int c = 0; c < K; ++c) {
            for (int el = 0; el < N; ++el) {
                hmat_ans.setat(r, c, hmat_ans(r,c) + hmat1(r, el) * hmat2(el, c));
            }
        }
    }
    auto hmat_prod = hmat1 * hmat2;

    EXPECT_EQ(hmat_prod, hmat_ans);
    EXPECT_EQ(hmat_ans.nR(), hmat_prod.nR());
    EXPECT_EQ(hmat_ans.nC(), hmat_prod.nC());

}

TEST(Matrix, Reshape)
{
    HostMatrix<float> mat1(2, 3);
    mat1 = 1, 2, 3, 
           4, 5, 6;

    HostMatrix<float> mat2(3, 2);
    mat2 = 1, 5,
           4, 3,
           2, 6;

    EXPECT_EQ(mat2.reshape(2, 3), mat1);
}

TEST(Matrix, TransposeReshape)
{
    HostMatrix<float> mat(2, 3);
    mat = 1, 3, 4, 5, 7, 9;

    HostMatrix<float> ans(2, 3);
    ans = 1, 4, 7, 3, 5, 9;

    auto mat_trans = mat.transpose();
    auto mat_res = mat_trans.reshape(2, 3);

    EXPECT_EQ(ans, mat_res);
}

TEST(Matrix, ReshapeTranspose)
{
    HostMatrix<float> mat(2, 3);
    mat = 1, 3, 4, 5, 7, 9;

    HostMatrix<float> ans(2, 3);
    ans = 1, 5, 3, 7, 4, 9;

    auto mat_trans = mat.reshape(3, 2).transpose();

    EXPECT_EQ(ans, mat_trans);
}

TEST(Matrix, TransposeReshapeTranspose)
{
    HostMatrix<float> mat(2, 3);
    mat = 1, 3, 4, 5, 7, 9;

    HostMatrix<float> ans(3, 2);
    ans = 1, 3, 4, 5, 7, 9;

    auto mat_trans = mat.transpose().reshape(2, 3).transpose();

    EXPECT_EQ(ans, mat_trans);
}

TEST(Matrix, ReshapeTransposeReshape)
{
    HostMatrix<float> mat(2, 3);
    mat = 1, 3, 4, 5, 7, 9;

    HostMatrix<float> ans(2, 3);
    ans = 1, 4, 7, 3, 5 ,9;

    auto mat_trans = mat.reshape(2, 3).transpose().reshape(2, 3);

    EXPECT_EQ(ans, mat_trans);
}

TEST(Matrix, TraceEqual)
{
    HostMatrix<float> mat(2, 2);
    mat = 1, 2, 3, 4;
    EXPECT_EQ(mat.trace(), 5);
}
TEST(Matrix, TraceTall)
{
    HostMatrix<float> mat(3, 2);
    mat = 1, 2, 3, 4, 5, 6;
    EXPECT_EQ(mat.trace(), 5);
}
TEST(Matrix, TraceWide)
{
    HostMatrix<float> mat(2, 3);
    mat = 1, 2, 3, 4, 5, 6;
    EXPECT_EQ(mat.trace(), 6);
}
int main(int argc, char **argv)
{
    std::cout << "Running test" << std::endl;
    ::testing::InitGoogleTest(&argc, argv);

    return RUN_ALL_TESTS();
}
